#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <vector> 
#include <algorithm> 

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#define SEQ_SIZE 9
#define PATTERN_SIZE 4


__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void multiplyBy2(int* data, unsigned int n) {
	unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x; 
	if (tid < n) { 
		data[tid] = 2 * data[tid]; 
	} 
}

template<typename T>
std::vector<T>* getUniqueValues(std::vector<T>* input) {
	std::vector<T>* uniqueValues = new std::vector<T>(*input);
	std::sort(uniqueValues->begin(), uniqueValues->end());
	auto ip = std::unique(uniqueValues->begin(), uniqueValues->end());
	auto begin = uniqueValues->begin();
	uniqueValues->resize(std::distance(begin, ip));
	return uniqueValues;
}

template<typename T>
thrust::host_vector<T>* getHostVector(std::vector<T>* input) {
	thrust::host_vector<T>* host_vector = new thrust::host_vector<T>();
	for (auto it = input->begin(); it != input->end(); ++it) {
		host_vector->push_back(*it);
	}
	return host_vector;
}

unsigned long long variations_without_repetitions_count(int n, int k) {
	if (k > n) {
		return 1;
	}

	unsigned long long result = 1;
	for (int i = n; i > n - k; i--) {
		result *= i;
	}

	return result;
}


int main()
{  
	// 124353621
    //const int seq[SEQ_SIZE] = { 1,2, 4, 3, 5, 3, 6, 2, 1 };
    //const char pattern[PATTERN_SIZE] = { 'a', 'b', 'b', 'a' };
	std::vector<int> seq = { 1,2, 4, 3, 5, 3, 6, 2, 1 };
	std::vector<char> pattern = { 'a', 'b', 'b', 'a' };

	thrust::host_vector<char>* patternValues = getHostVector(getUniqueValues(&pattern));
	thrust::host_vector<int>* seqValues = getHostVector(getUniqueValues(&seq));

	thrust::device_vector<char>* devicePatternValues = new thrust::device_vector<char>();
	thrust::device_vector<int>* deviceSeqValues = new thrust::device_vector<int>();

	deviceSeqValues->resize(seqValues->size());

	*deviceSeqValues = *seqValues;

	multiplyBy2 <<< 1, 10 >>> (deviceSeqValues->data().get(), deviceSeqValues->size());

	*seqValues = *deviceSeqValues;

	for (auto it = seqValues->begin(); it != seqValues->end(); ++it) {
		std::cout << *it;
	}

	//std::cout << variations_without_repetitions_count(15, 12);

	free(seqValues);
	free(patternValues);
    //// Add vectors in parallel.
    //hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addWithCuda failed!");
    //    return 1;
    //}

    //printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
    //    c[0], c[1], c[2], c[3], c[4]);

    //// hipDeviceReset must be called before exiting in order for profiling and
    //// tracing tools such as Nsight and Visual Profiler to show complete traces.
    //cudaStatus = hipDeviceReset();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceReset failed!");
    //    return 1;
    //}

    return 0;
}