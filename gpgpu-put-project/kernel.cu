#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <vector> 
#include <algorithm> 

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


#define BLOCK_SIZE 1024


__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void multiplyBy2(int* data, unsigned int n) {
	unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x; 
	if (tid < n) { 
		data[tid] = 2 * data[tid]; 
	} 
}

template<typename T>
std::vector<T>* getUniqueValues(std::vector<T>* input) {
	std::vector<T>* uniqueValues = new std::vector<T>(*input);
	std::sort(uniqueValues->begin(), uniqueValues->end());
	auto ip = std::unique(uniqueValues->begin(), uniqueValues->end());
	auto begin = uniqueValues->begin();
	uniqueValues->resize(std::distance(begin, ip));
	return uniqueValues;
}

template<typename T>
thrust::host_vector<T>* getHostVector(std::vector<T>* input) {
	thrust::host_vector<T>* host_vector = new thrust::host_vector<T>();
	for (auto it = input->begin(); it != input->end(); ++it) {
		host_vector->push_back(*it);
	}
	return host_vector;
}

__host__ __device__ void variations_without_repetitions_count(int n, int k, unsigned long long* result) {
	if (k > n) {
		*result = 1;
		return;
	}

	*result = 1;
	for (int i = n; i > n - k; i--) {
		*result *= i;
	}
}

__host__ __device__ void variation(int n, int k, int variationNumber, int* result) {
	bool* isTaken = new bool[n];
	for (int i = 0; i < n; i++) {
		isTaken[i] = false;
	}
	for (int x = 0; x < k ; x++) {
		unsigned long long v = 0;
		variations_without_repetitions_count(n - x - 1, k - x - 1, &v);
		auto t = variationNumber / v;
		int searchedPosition = -1;
		int realPosition = 0;
		for (int i = 0; i < n; i++) {
			if (!isTaken[i]) {
				searchedPosition++;
				if (t == searchedPosition) {
					realPosition = i;
					break;
				}
			}
			
		}
		isTaken[realPosition] = true;
		result[x] = realPosition;
		variationNumber %= v;
	}
}

__global__ void findSubstitution(
	char* patternValues, int patternValuesSize, 
	int* seqValues, int seqValuesSize, 
	char* pattern, int patternSize,
	int* seq, int seqSize,
	int* result, unsigned long long variationCount) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index > variationCount) return;
	
	int* variationResult = new int[patternValuesSize];
	variation(seqValuesSize, patternValuesSize, index, variationResult);

	int* patternWithValues = new int[patternSize];
	for (int i = 0; i < patternValuesSize; i++) {
		for (int j = 0; j < patternSize; j++) {
			if (patternValues[i] == pattern[j]) {
				patternWithValues[j] = seqValues[variationResult[i]];
			}
		}
	}
	
	int patternIndex = 0;
	for (int i = 0; i < seqSize && patternIndex < patternSize; i++) {
		if (seq[i] == patternWithValues[patternIndex]) {
			patternIndex++;
		}
	}
	if (patternIndex == patternSize) {
		result[index] = 1;
	}
	else {
		result[index] = 0;
	}
}

int main()
{  
	std::vector<int> seq = { 1,2, 4, 3, 5, 3, 6, 2, 1 };
	std::vector<char> pattern = { 'a', 'b', 'b', 'a' };

	thrust::host_vector<char>* patternValues = getHostVector(getUniqueValues(&pattern));
	thrust::host_vector<char>* thrustPattern = getHostVector(&pattern);
	thrust::host_vector<int>* seqValues = getHostVector(getUniqueValues(&seq));
	thrust::host_vector<int>* thrustSeq = getHostVector(&seq);
	thrust::host_vector<int>* result = new thrust::host_vector<int>();
	

	thrust::device_vector<char>* devPatternValues = new thrust::device_vector<char>();
	thrust::device_vector<char>* devThrustPattern = new thrust::device_vector<char>();
	thrust::device_vector<int>* devSeqValues = new thrust::device_vector<int>();
	thrust::device_vector<int>* devThrustSeq = new thrust::device_vector<int>();
	thrust::device_vector<int>* devResult = new thrust::device_vector<int>();

	unsigned long long variationCount = 0;
	variations_without_repetitions_count(seqValues->size(), patternValues->size(), &variationCount);
	int gridSize = variationCount / BLOCK_SIZE;
	if (gridSize < 1) {
		gridSize = 1;
	}
	

	devPatternValues->resize(patternValues->size());
	devThrustPattern->resize(thrustPattern->size());
	devSeqValues->resize(seqValues->size());
	devThrustSeq->resize(thrustSeq->size());

	result->resize(variationCount);
	devResult->resize(variationCount);

	*devPatternValues = *patternValues;
	*devThrustPattern = *thrustPattern;
	*devSeqValues = *seqValues;
	*devThrustSeq = *thrustSeq;
	*devResult = *result;

	findSubstitution <<< gridSize, BLOCK_SIZE >>> (
		devPatternValues->data().get(), devPatternValues->size(),
		devSeqValues->data().get(), devSeqValues->size(),
		devThrustPattern->data().get(), devThrustPattern->size(),
		devThrustSeq->data().get(), devThrustSeq->size(),
		devResult->data().get(), variationCount);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		std::cout << "cuda error: " << hipGetErrorString(err) << std::endl;
		return 1;
	}


	*result = *devResult;

	for (int i = 0; i < result->size(); i++) {
		if ((*result)[i] != 0) {
			int* variationResult = new int[patternValues->size()];
			variation(seqValues->size(), patternValues->size(), i, variationResult);
			for (int i = 0; i < patternValues->size(); i++) {
				std::cout << (*patternValues)[i] << "=" << (*seqValues)[variationResult[i]] << " ";
			}
			std::cout << std::endl;
		}
	}


    return 0;
}